#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "CudaUtils.cuh"
#include "ForceKernels.cuh"
#include "IntegrationKernels.cuh"
#include "SimulationKernels.h"
#include "Constants.cuh"

// Keep device constant values - good for performance
__constant__ double G_CUDA;
__constant__ double SOFTENING_SQUARED;
__constant__ double PI_CUDA;

// Simplify helper function declarations
std::string getDevicePropertiesAsync(hipStream_t stream);
void releaseGPUResources(hipStream_t computeStream, hipStream_t dataStream, hipStream_t setupStream,
                        hipEvent_t computeDone, hipEvent_t dataReady, hipEvent_t energyCalculated,
                        void* d_posMass, void* d_vel, void* d_accel, void* d_totalEnergy, void* d_accelOld);

// Main entry point for GPU simulation
extern "C" void runSimulationOnGPU(Particles& particles, 
                                  IntegrationMethod method,
                                  ForceMethod forceMethod,
                                  double dt, int steps, int stepFreq,
                                  OutputData& outputData) {
    
    int n = particles.n;
    std::cout << "Starting GPU simulation with " << n << " particles..." << std::endl;
    
    // Check if CUDA is available
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess || deviceCount == 0) {
        std::cerr << "CUDA error: " << (error != hipSuccess ? 
            hipGetErrorString(error) : "No CUDA-capable devices found") << std::endl;
        throw std::runtime_error("CUDA initialization failed");
    }
    
    // Get current device information
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    if (props.major < 2) {
        std::cerr << "Error: This simulation requires at least compute capability 2.0" << std::endl;
        throw std::runtime_error("GPU too old for simulation");
    }
    
    std::cout << "Using CUDA device: " << props.name << " (CC " << props.major << "." << props.minor << ")" << std::endl;
    
    // Create CUDA streams for overlapping operations
    hipStream_t computeStream = nullptr, dataStream = nullptr, setupStream = nullptr;
    hipEvent_t computeDone = nullptr, dataReady = nullptr, energyCalculated = nullptr;
    
    // Simplify stream and event creation with a try-catch block
    try {
        gpuErrchk(hipStreamCreateWithFlags(&computeStream, hipStreamNonBlocking));
        gpuErrchk(hipStreamCreateWithFlags(&dataStream, hipStreamNonBlocking));
        gpuErrchk(hipStreamCreateWithFlags(&setupStream, hipStreamNonBlocking));
        gpuErrchk(hipEventCreate(&computeDone));
        gpuErrchk(hipEventCreate(&dataReady));
        gpuErrchk(hipEventCreate(&energyCalculated));
    } catch (const std::exception& e) {
        releaseGPUResources(computeStream, dataStream, setupStream, 
                          computeDone, dataReady, energyCalculated, 
                          nullptr, nullptr, nullptr, nullptr, nullptr);
        throw;
    }
    
    // Start device property query in separate stream
    std::string deviceInfo = getDevicePropertiesAsync(setupStream);
    
    // Allocate device memory - simplified approach
    double4 *d_posMass = nullptr, *d_vel = nullptr, *d_accel = nullptr, *d_accelOld = nullptr;
    double *d_totalEnergy = nullptr;
    bool usingAsyncAlloc = false;
    
    try {
        #if CUDART_VERSION >= 11020
        // Check if device supports async allocations
        int driverVersion = 0;
        hipDriverGetVersion(&driverVersion);
        
        if (driverVersion >= 11020 && props.major >= 7) {
            usingAsyncAlloc = true;
            std::cout << "Using asynchronous memory operations" << std::endl;
            
            // Try asynchronous allocations
            hipError_t err = hipMallocAsync(&d_posMass, n * sizeof(double4), computeStream);
            if (err == hipSuccess) {
                gpuErrchk(hipMallocAsync(&d_vel, n * sizeof(double4), computeStream));
                gpuErrchk(hipMallocAsync(&d_accel, n * sizeof(double4), computeStream));
                gpuErrchk(hipMallocAsync(&d_totalEnergy, sizeof(double), computeStream));
                
                if (method == IntegrationMethod::VELOCITY_VERLET) {
                    gpuErrchk(hipMallocAsync(&d_accelOld, n * sizeof(double4), computeStream));
                }
            } else {
                usingAsyncAlloc = false;
            }
        }
        #endif
        
        // Fall back to standard allocation if needed
        if (!usingAsyncAlloc) {
            gpuErrchk(hipMalloc(&d_posMass, n * sizeof(double4)));
            gpuErrchk(hipMalloc(&d_vel, n * sizeof(double4)));
            gpuErrchk(hipMalloc(&d_accel, n * sizeof(double4)));
            gpuErrchk(hipMalloc(&d_totalEnergy, sizeof(double)));
            
            if (method == IntegrationMethod::VELOCITY_VERLET) {
                gpuErrchk(hipMalloc(&d_accelOld, n * sizeof(double4)));
            }
        }
    } catch (const std::exception& e) {
        releaseGPUResources(computeStream, dataStream, setupStream, 
                          computeDone, dataReady, energyCalculated, 
                          d_posMass, d_vel, d_accel, d_totalEnergy, d_accelOld);
        throw;
    }
    
    // Calculate average mass for adaptive methods
    double avgMass = 0.0;
    if (forceMethod == ForceMethod::ADAPTIVE_MUTUAL) {
        for (int i = 0; i < n; ++i) {
            avgMass += particles.posMass[i].w;
        }
        avgMass /= n;
    }
    
    // Copy initial particle data to device
    gpuErrchk(hipMemcpyAsync(d_posMass, particles.posMass, 
                          n * sizeof(double4), hipMemcpyHostToDevice, computeStream));
    gpuErrchk(hipMemcpyAsync(d_vel, particles.vel, 
                          n * sizeof(double4), hipMemcpyHostToDevice, computeStream));
    
    // Record when data is ready
    gpuErrchk(hipEventRecord(dataReady, computeStream));
    
    // Wait for setup to complete and display info
    gpuErrchk(hipStreamSynchronize(setupStream));
    std::cout << deviceInfo << std::endl;
    
    // Determine optimal block size
    size_t sharedMemSize = sizeof(double4) * 32;
    int blockSize = determineOptimalBlockSize(n, sharedMemSize);
    dim3 blocks = calculateGrid(n, blockSize);
    
    std::cout << "Using block size: " << blockSize << ", grid size: " << blocks.x << std::endl;
    
    // Initialize constants
    initializeConstants();
    
    // Store static data
    storeStaticDataKernel<<<blocks, blockSize, 0, dataStream>>>(
        d_posMass, outputData.getDeviceStaticDataPtr(), n);
    gpuErrchk(hipGetLastError());
    
    // Wait for data to be on device
    gpuErrchk(hipStreamWaitEvent(computeStream, dataReady));
    
    // Calculate initial forces
    sharedMemSize = blockSize * sizeof(double4);
    launchForceKernel(forceMethod, blocks, blockSize, sharedMemSize, computeStream, 
                     d_posMass, d_accel, avgMass, n);
    gpuErrchk(hipGetLastError());
    
    // Calculate initial energy
    gpuErrchk(hipMemsetAsync(d_totalEnergy, 0, sizeof(double), computeStream));
    size_t energySharedMem = blockSize * sizeof(double) * 5;
    computeTotalEnergyKernel<<<blocks, blockSize, energySharedMem, computeStream>>>(
        d_posMass, d_vel, d_totalEnergy, n);
    gpuErrchk(hipGetLastError());
    
    // Record energy calculation completion
    gpuErrchk(hipEventRecord(energyCalculated, computeStream));
    gpuErrchk(hipStreamWaitEvent(dataStream, energyCalculated));
    
    // Store initial state
    storeParticleStateKernel<<<blocks, blockSize, 0, dataStream>>>(
        d_posMass, d_vel, 
        outputData.d_particleData, outputData.d_systemData,
        d_totalEnergy,
        n, 0, 0.0);
    
    // Main simulation loop
    int printCounter = 0;
    size_t timeIdx = 1;
    
    for (int step = 1; step <= steps; ++step) {
        double time = step * dt;
        
        performGPUIntegrationStep(
            d_posMass, d_vel, d_accel, d_accelOld,
            method, forceMethod,
            dt, avgMass, n,
            blocks, blockSize, sharedMemSize,
            computeStream);
        
        // Store state periodically
        if (++printCounter >= stepFreq) {
            gpuErrchk(hipMemsetAsync(d_totalEnergy, 0, sizeof(double), computeStream));
            computeTotalEnergyKernel<<<blocks, blockSize, energySharedMem, computeStream>>>(
                d_posMass, d_vel, d_totalEnergy, n);
            gpuErrchk(hipGetLastError());
            
            gpuErrchk(hipEventRecord(energyCalculated, computeStream));
            gpuErrchk(hipStreamWaitEvent(dataStream, energyCalculated));
            
            storeParticleStateKernel<<<blocks, blockSize, 0, dataStream>>>(
                d_posMass, d_vel,
                outputData.d_particleData, outputData.d_systemData,
                d_totalEnergy,
                n, timeIdx, time);
            
            int progressPercent = (step * 100) / steps;
            printf("\rProgress: %d%%", progressPercent);
            fflush(stdout);
            
            timeIdx++;
            printCounter = 0;
        }
    }
    
    // Final synchronization
    gpuErrchk(hipEventRecord(computeDone, computeStream));
    gpuErrchk(hipEventSynchronize(computeDone));
    printf("\rProgress: 100%%\n");
    
    // Transfer data
    gpuErrchk(hipMemcpyAsync(outputData.h_particleData, outputData.d_particleData, 
                           outputData.numParticles * outputData.numTimeSteps * outputData.valuesPerParticle * sizeof(double), 
                           hipMemcpyDeviceToHost, dataStream));
    
    gpuErrchk(hipMemcpyAsync(outputData.h_systemData, outputData.d_systemData, 
                           outputData.numTimeSteps * outputData.valuesPerSystem * sizeof(double), 
                           hipMemcpyDeviceToHost, dataStream));
    
    gpuErrchk(hipMemcpyAsync(outputData.h_staticData, outputData.d_staticData, 
                           outputData.numParticles * outputData.valuesPerStatic * sizeof(double), 
                           hipMemcpyDeviceToHost, dataStream));
    
    gpuErrchk(hipMemcpyAsync(particles.posMass, d_posMass, n * sizeof(double4), 
                           hipMemcpyDeviceToHost, dataStream));
    gpuErrchk(hipMemcpyAsync(particles.vel, d_vel, n * sizeof(double4), 
                           hipMemcpyDeviceToHost, dataStream));
    
    // Wait for all transfers to complete
    gpuErrchk(hipStreamSynchronize(dataStream));
    
    // Free resources using the appropriate method
    #if CUDART_VERSION >= 11020
    if (usingAsyncAlloc) {
        if (d_posMass) hipFreeAsync(d_posMass, computeStream);
        if (d_vel) hipFreeAsync(d_vel, computeStream);
        if (d_accel) hipFreeAsync(d_accel, computeStream);
        if (d_totalEnergy) hipFreeAsync(d_totalEnergy, computeStream);
        if (method == IntegrationMethod::VELOCITY_VERLET && d_accelOld)
            hipFreeAsync(d_accelOld, computeStream);
        hipStreamSynchronize(computeStream);
    } else {
    #endif
        if (d_posMass) hipFree(d_posMass);
        if (d_vel) hipFree(d_vel);
        if (d_accel) hipFree(d_accel);
        if (d_totalEnergy) hipFree(d_totalEnergy);
        if (method == IntegrationMethod::VELOCITY_VERLET && d_accelOld)
            hipFree(d_accelOld);
    #if CUDART_VERSION >= 11020
    }
    #endif
    
    // Cleanup events and streams
    if (computeDone) hipEventDestroy(computeDone);
    if (dataReady) hipEventDestroy(dataReady);
    if (energyCalculated) hipEventDestroy(energyCalculated);
    
    if (computeStream) hipStreamDestroy(computeStream);
    if (dataStream) hipStreamDestroy(dataStream);
    if (setupStream) hipStreamDestroy(setupStream);
    
    std::cout << "GPU simulation completed successfully." << std::endl;
}

// Simplified helper function implementations
std::string getDevicePropertiesAsync(hipStream_t stream) {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    
    std::stringstream ss;
    ss << "Compute capability: " << prop.major << "." << prop.minor << "\n"
       << "Global memory: " << prop.totalGlobalMem / (1024*1024) << " MB\n"
       << "Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB";
       
    return ss.str();
}

void releaseGPUResources(hipStream_t computeStream, hipStream_t dataStream, hipStream_t setupStream,
                        hipEvent_t computeDone, hipEvent_t dataReady, hipEvent_t energyCalculated,
                        void* d_posMass, void* d_vel, void* d_accel, void* d_totalEnergy, void* d_accelOld) {
    
    // Free device memory
    if (d_posMass) hipFree(d_posMass);
    if (d_vel) hipFree(d_vel);
    if (d_accel) hipFree(d_accel);
    if (d_totalEnergy) hipFree(d_totalEnergy);
    if (d_accelOld) hipFree(d_accelOld);
    
    // Destroy events
    if (computeDone) hipEventDestroy(computeDone);
    if (dataReady) hipEventDestroy(dataReady);
    if (energyCalculated) hipEventDestroy(energyCalculated);
    
    // Destroy streams
    if (computeStream) hipStreamDestroy(computeStream);
    if (dataStream) hipStreamDestroy(dataStream);
    if (setupStream) hipStreamDestroy(setupStream);
}